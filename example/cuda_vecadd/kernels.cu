#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernels.hpp"

#define CHECK_CUDA(func) { \
    hipError_t err = (func); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error @ %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        abort(); \
    } \
}

__global__ void init_kernel(double *A, double *B, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        A[tid] = tid;
        B[tid] = 2 * tid;
    }
}

void initialize_device_arrays(double *dA, double *dB, int N) {
    int threads_per_block = 128;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
    init_kernel<<<blocks_per_grid, threads_per_block>>>(dA, dB, N);
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipGetLastError());
}

__global__ void vecadd_kernel(double *A, double *B, double *C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        C[tid] = A[tid] + B[tid];
    }
}

void gpu_vector_sum(double *dA, double *dB, double *dC, int start, int end) {
    int N = end - start;
    int threads_per_block = 128;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    vecadd_kernel<<<blocks_per_grid, threads_per_block>>>(dA + start, dB + start,
            dC + start, N);
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipGetLastError());
}
